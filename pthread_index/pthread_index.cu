#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils.h"

__global__ void printThreadIndexKernel(float *A, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    printf("thread_id(%d, %d) block_id(%d, %d)\
            coordinate(%d, %d) global index %2d ival %f\n", 
            threadIdx.x, threadIdx.y,
            blockIdx.x, blockIdx.y, 
            ix, iy, idx, A[idx]);
}
int main(int argc, char** argv)
{
    initDevice(0);
    int nx = 8, ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    //Malloc
    float *A_host = (float*)malloc(nBytes);
    initialData(A_host, nxy);
    printMatrix(A_host, nx, ny);

    //hipMalloc
    float *A_dev = nullptr;
    CHECKCUDAERR(hipMalloc((void**)&A_dev, nBytes));

    hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice);

    dim3 block(4, 2);
    dim3 grid((nx - 1) / block.x + 1, (ny - 1) / block.y + 1);

    printThreadIndexKernel<<<grid, block>>>(A_dev, nx, ny);

    CHECKCUDAERR(hipDeviceSynchronize());
    
    hipFree(A_dev);
    free(A_host);

    hipDeviceReset();
    return 0;
}
